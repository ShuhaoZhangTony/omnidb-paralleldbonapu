#ifndef TEST_COP_CU
#define TEST_COP_CU

#include "TestAll.cu"
#include "MyThreadPool.cu"

struct ws_sort
{
	Record *Rout;
	int startID;
	int endID;
	int ID;
};

HANDLE  gpuInUseMutex;                 


int compareInt (const void * a, const void * b)
{
  return ( ((int*)a) - ((int*)b));
}


DWORD WINAPI tp_sort( LPVOID lpParam ) 
{ 
	WaitForSingleObject( gpuInUseMutex, INFINITE );

	ws_sort* pData;
	pData = (ws_sort*)lpParam;
	int ID=pData->ID;
	if(ID%2==0)
	{
		printf("gpu sorted \n");
		testSort(16*1024*1024);	
		printf("gpu eneded\n");
	}
	else
	{
		printf("cpu sorted \n");
		int len=16*1024*1024;
		int *data=(int*)malloc(sizeof(int)*len);
		int i=0;
		for(i=0;i<len;i++)
			data[i]=rand();
		qsort(data,len,sizeof(int),compareInt);
		printf("cpu ended \n");
	}
	ReleaseMutex( gpuInUseMutex );

	return 0;
} 

void testCoProcessing(int argc, char** argv)
{
	gpuInUseMutex = CreateMutex( NULL, FALSE, NULL );  // Cleared 


	MyThreadPool *pool=(MyThreadPool*)malloc(sizeof(MyThreadPool));
	int numThread=4;
	pool->create(numThread);
	int i=0;
	ws_sort** pData=(ws_sort**)malloc(sizeof(ws_sort*)*numThread);
	for( i=0; i<numThread; i++ )
	{
		// Allocate memory for thread data.
		pData[i] = (ws_sort*) HeapAlloc(GetProcessHeap(),
				HEAP_ZERO_MEMORY, sizeof(ws_sort));

		if( pData[i]  == NULL )
			ExitProcess(2);

		// Generate unique data for each thread.
		pData[i]->ID=i;
		pool->assignParameter(i, pData[i]);
		pool->assignTask(i, tp_sort);
	}
	pool->run();
	for(i=0;i<numThread;i++)
		HeapFree(GetProcessHeap(),0, pData[i]);
	free(pData);
	pool->destory();
	printf("co-processing okay");
	CloseHandle( gpuInUseMutex );

	
}

#endif